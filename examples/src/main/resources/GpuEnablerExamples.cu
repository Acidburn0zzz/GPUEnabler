
#include <hip/hip_runtime.h>

extern "C"
// another simple test kernel
__global__ void multiplyBy2(int *size, int *in, int *out) {
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < *size) {
        out[ix] = in[ix] * 2;
    }
}


extern "C"
// test reduce kernel that sums elements
__global__ void sum(int *size, int *input, int *output, int *stage, int *totalStages) {
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    const int jump = 64 * 256;
    // if (ix == 0) printf("size: %d stage : %d totalStages : %d \n",*size, *stage, *totalStages);
    if (*stage == 0) {
        if (ix < *size) {
            assert(jump == blockDim.x * gridDim.x);
            int result = 0;
            for (long i = ix; i < *size; i += jump) {
                result += input[i];
            }
            input[ix] = result;
        }
    } else if (ix == 0) {
        const long count = (*size < (long)jump) ? *size : (long)jump;
        int result = 0;
        for (long i = 0; i < count; ++i) {
            result += input[i];
        }
        output[0] = result;
    }
}


extern "C"
// test reduce kernel that sums elements
__global__ void sum1(int *size, int *input, int *output) {
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    if (ix == 0) {
        int result = 0;
        for (long i = ix; i < *size; i++) {
            result += input[i];
        }
        output[0] = result;
    }
}


extern "C"
__global__ void add(int n, long *a, long *b, long *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        sum[i] = a[i] + b[i];
        printf("CUDA KERNEL ADD %ld + %ld = %ld \n",a[i],b[i],sum[i]);
    }

}

extern "C"
__global__ void mul(int n, long *a, long *b, long *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        sum[i] = a[i] * b[i];
        printf("CUDA KERNEL MUL %ld * %ld = %ld \n",a[i],b[i],sum[i]);
    }

}

extern "C"
__global__ void arrayTest(int n, long *a, long *b, long *arr,long *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i == 0) {
       printf("In ArrayTest n=%d a=%p b=%p out=%p arr=%p\n",n,a,b,sum,arr);
    }

    if (i<n)
    {
        sum[i] = a[i] * b[i];
        printf("CUDA KERNEL ArrayTest a=%ld b=%ld sum=%ld arr=[%ld %ld %ld]\n",a[i],b[i],sum[i],arr[i*3],arr[(i*3)+1],arr[(i*3)+2]);
    }

}
